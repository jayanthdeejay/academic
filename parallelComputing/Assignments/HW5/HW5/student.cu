#include "hip/hip_runtime.h"
#include "utils.h"
#include "reference_calc.cpp"

#define BLOCK_SIZE_MAX 512
#define GRID_SIZE_MAX 512
#define NUMBER_OF_ELEMS_PER_THREAD 16

__global__
void histogramKernel(const unsigned int* const d_In, unsigned int* const d_Out, int numVals, unsigned int valsOffset, unsigned int numBins) {
    extern __shared__ unsigned int s_histogramKernel_Out[];
    int threadsPerBlock = blockDim.x * blockDim.y;
    int threadsPerGrid = threadsPerBlock * gridDim.x * gridDim.y;
    int blockId = blockIdx.x + (blockIdx.y * gridDim.x);
    int threadId = threadIdx.x + (threadIdx.y * blockDim.x);
    for (int i = 0; i < (numBins / threadsPerBlock); i++) {
        int _index =
        i * threadsPerBlock + threadId;
        if (_index < numBins) {
            s_histogramKernel_Out[_index] =0;
        }
    }
    __syncthreads();
    int myId = (blockId * threadsPerBlock) + threadId;
    for (int _step = 0; _step < NUMBER_OF_ELEMS_PER_THREAD; _step++) {
        int _myTrueId =
        myId + _step * threadsPerGrid;
        if ( (_myTrueId + valsOffset) >= numVals ) {
            break;
        }
        else {
            unsigned int _in =
            d_In[_myTrueId];
            atomicAdd(&(s_histogramKernel_Out[_in]), 1);
        }
    }
    __syncthreads();
    for (int i = 0; i < (numBins / threadsPerBlock); i++) {
        int _index = i * threadsPerBlock + threadId;
        if (_index < numBins) {
            atomicAdd(&(d_Out[_index]), s_histogramKernel_Out[_index]);
        }
    }
}

void computeHistogram(const unsigned int* const d_In, unsigned int* const d_Out, const unsigned int numBins, const unsigned int numElems) {
    unsigned int _numElemsProcessed = 0;
    dim3 _block(BLOCK_SIZE_MAX);
    while (_numElemsProcessed < numElems) {
        int numElemGroupsLeft =
        (numElems - _numElemsProcessed - 1) / NUMBER_OF_ELEMS_PER_THREAD + 1;
        int _gridSize = (numElemGroupsLeft - 1) / BLOCK_SIZE_MAX + 1;
        _gridSize = _gridSize < GRID_SIZE_MAX ? _gridSize : GRID_SIZE_MAX;
        dim3 _grid(_gridSize);
        histogramKernel<<<_grid, _block, (numBins * sizeof(unsigned int))>>> (&d_In[_numElemsProcessed],d_Out,numElems,_numElemsProcessed,numBins);
        _numElemsProcessed +=_gridSize * BLOCK_SIZE_MAX * NUMBER_OF_ELEMS_PER_THREAD;
}
hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}