#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include<stdio.h>
#include<cmath>
__device__ inline int getidx(){
    return threadIdx.x+threadIdx.y*blockDim.x+blockIdx.x*blockDim.x*blockDim.y;};
template <typename T>
__device__ inline int lessEq(const T a, const T b){
    return (int)(a<=b);
};
template <typename T>
__device__ inline int moreEq(const T a, const T b){
    return (int)(a>=b);
};
template <typename S, int (*T) (const S, const S)>
__global__ void bitonic_sort_ (S* const in, S* const in2, const int len, const int inc, const int inr){
    int idx = getidx();
    if (idx >= len) return;
    int updown = 0;
    S up, down;
    S up_, down_;
    int pass=0;
    {
        updown = (idx/inc) % 2;
        up = in[idx];
        up_ = in2[idx];
        if (idx % (inr*2) >= inr) return;
        down = in[idx+inr];
        down_= in2[idx+inr];
        pass = (T(up,down)==updown);
        if (pass)  return;
        else {
            in[idx]=down;
            in2[idx]=down_;
            in[idx+inr]=up;
            in2[idx+inr]=up_;
        }
    }
    return;
}
template <typename S, int (*T) (const S, const S)>
int bitonic_sort (S* const in, S* const in2,
const int len, const dim3& gsize, const dim3& bsize) {
    int cnt =0;
    for (int inc = 2; inc<=len; inc=inc*2) {
        for (int inr = inc/2; inr>=1; inr=inr/2) {
            bitonic_sort_ <S, T> <<<gsize,bsize>>>(in, in2, len, inc, inr);
        }
        cnt++;
    }
    return cnt;
    }
template <typename S>
__global__ void setdummy(const S* const in, S* const out, const S dummy, const int len1, const int len2) {
    int idx=getidx();
    S val;
    if (idx >= len2) return;
    if (idx < len1) val = in[idx];
    else val =dummy;
    out[idx] = val;
    return;
}
template <typename S>
unsigned int pad( const S* const in, S* out, unsigned int len, S dummy) {
    unsigned int exp = (unsigned int)log2((float)len)+1;
    unsigned int acc = (unsigned int)exp2((float)exp);
    printf("padded %d elements to %d\n", len, acc);
    return acc;
}

void your_sort(unsigned int* const d_inputVals, unsigned int* const d_inputPos, unsigned int* const d_outputVals, unsigned int* const d_outputPos, const size_t numElems) {
    unsigned int *d_tmp, *d_tmp2;
    unsigned int acc = pad <unsigned> (d_inputVals, d_tmp, numElems, (unsigned)(-1));
    pad <unsigned> (d_inputPos, d_tmp2, numElems, (unsigned)(-1));
    dim3 bsize2(64,16,1);
    int numGrids=(int)acc/(64*16)+1;
    dim3 gsize2(numGrids,1,1);
    checkCudaErrors(hipMalloc((void**) &d_tmp, sizeof(unsigned int)*acc));
    checkCudaErrors(hipMalloc((void**) &d_tmp2, sizeof(unsigned int)*acc));
    setdummy <unsigned int> <<<gsize2, bsize2>>> (d_inputVals, d_tmp,(unsigned int)(-1), numElems, acc);
    setdummy <unsigned int> <<<gsize2, bsize2>>> (d_inputPos, d_tmp2,(unsigned int)(-1), numElems, acc);
    int cnt =0;
    cnt = bitonic_sort<unsigned int, moreEq<unsigned int> > (d_tmp, d_tmp2, (int)acc,  gsize2, bsize2);
    setdummy <unsigned int> <<<gsize2, bsize2>>> (d_tmp,d_outputVals,0, acc, numElems);
    setdummy <unsigned int> <<<gsize2, bsize2>>> (d_tmp2,d_outputPos,0, acc, numElems);
    checkCudaErrors(hipFree(d_tmp));
    checkCudaErrors(hipFree(d_tmp2));
    printf("Total length is %d, total iteration finished is %d\n", (int)acc, cnt);
}