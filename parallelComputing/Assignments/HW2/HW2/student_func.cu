#include "hip/hip_runtime.h"

#include "utils.h"

__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{

  // numRows=313 && numCols=557
    int X=blockIdx.x * blockDim.x + threadIdx.x;
    int Y=blockIdx.y * blockDim.y + threadIdx.y;
    if ( X >= numCols || Y >= numRows ) {
        return;
    }
    int half   = filterWidth / 2;
    float blur   = 0.0f;
    int width  = numCols - 1;
    int height = numRows - 1;
    
    for (int i = -half; i <= half; i++) {
        for (int j = -half; j <= half; j++) {
            int h=min(max(Y+i,0),height);
            int w=min(max(X+j,0),width);
            int idx=h*numCols+w;
            float pixel=static_cast<float>(inputChannel[idx]);
            idx=(i+half)+(filterWidth*(j+half));
            float weight=filter[idx];
            blur+=pixel*weight;
        }
    }
    
    outputChannel[X+numCols*Y] = static_cast<unsigned char>(blur);
}   

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
    int X=blockIdx.x * blockDim.x + threadIdx.x;
    int Y=blockIdx.y * blockDim.y + threadIdx.y;
    if ( X >= numCols || Y >= numRows ) {
        return;
    }
    uchar4 rgba = inputImageRGBA[X+numCols*Y];
    redChannel[X+numCols*Y]=rgba.x;
    greenChannel[X+numCols*Y]=rgba.y;
    blueChannel[X+numCols*Y]=rgba.z;
}
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  //original
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_filter,  sizeof(float) * static_cast<float>(filterWidth)*static_cast<float>(filterWidth)));
  checkCudaErrors(hipMemcpy(d_filter, h_filter,sizeof(float) * static_cast<float>(filterWidth)*static_cast<float>(filterWidth), hipMemcpyHostToDevice));

}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{

  const dim3 blockSize(9,9,1);
  const dim3 gridSize(62,35,1);
  checkCudaErrors(hipMemcpy(d_inputImageRGBA, h_inputImageRGBA, sizeof(uchar4) * static_cast<int>(numRows)*static_cast<int>(numCols), hipMemcpyHostToDevice));
  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA,numRows,numCols,d_red,d_green,d_blue);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());    
  gaussian_blur<<<gridSize, blockSize>>>(d_red,d_redBlurred,numRows,numCols,d_filter,filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize, blockSize>>>(d_green,d_greenBlurred,numRows,numCols,d_filter,filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize, blockSize>>>(d_blue,d_blueBlurred,numRows,numCols,d_filter,filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
